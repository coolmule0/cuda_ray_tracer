#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include "vec3cu.h"
#include "raycu.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ color ray_color(const ray& r) {
    vec3 unit_direction = unit_vector(r.direction());
    auto a = 0.5*(unit_direction.y() + 1.0);
    return (1.0-a)*color(1.0, 1.0, 1.0) + a*color(0.5, 0.7, 1.0);
}

__global__ void render(color *fb, int max_x, int max_y,
                        vec3 camera_center, vec3 pixel00_loc, vec3 pixel_delta_u, vec3 pixel_delta_v) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;

    auto pixel_center = pixel00_loc + (i * pixel_delta_u) + (j * pixel_delta_v);
    auto ray_direction = pixel_center - camera_center;

    ray r(camera_center, ray_direction);
    fb[pixel_index] = ray_color(r);
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    vec3 camera_center = vec3(0,0,0);
    auto plane_dist = 1.0;
    auto viewport_height = 2.0;
    auto viewport_width = viewport_height * (double(nx)/ny);
    auto viewport_u = vec3(viewport_width, 0, 0);
    auto viewport_v = vec3(0, -viewport_height, 0);
    auto pixel_delta_u = viewport_u / nx;
    auto pixel_delta_v = viewport_v / ny;

    vec3 focal = vec3(0,0,1);
    vec3 horizontal = vec3(1,0,0);
    vec3 vertical = vec3(0,1,0);
    auto viewport_upper_left = camera_center
                             - vec3(0, 0, plane_dist) - viewport_u/2 - viewport_v/2;

    auto pixel00_loc = viewport_upper_left + 0.5 * (pixel_delta_u + pixel_delta_v);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny, camera_center, pixel00_loc, pixel_delta_u, pixel_delta_v);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));
}